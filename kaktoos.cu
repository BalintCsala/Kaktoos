
#include <hip/hip_runtime.h>
#include <cstdint>
#include <memory.h>
#include <cstdio>
#include <ctime>
#include <thread>
#include <vector>
#include <mutex>
#include <chrono>

#define RANDOM_MULTIPLIER 0x5DEECE66DULL
#define RANDOM_ADDEND 0xBULL
#define RANDOM_MASK ((1ULL << 48ULL) - 1ULL)

#ifndef FLOOR_LEVEL
#define FLOOR_LEVEL 63LL
#endif

#ifndef WANTED_CACTUS_HEIGHT
#define WANTED_CACTUS_HEIGHT 8ULL
#endif

#ifndef WORK_UNIT_SIZE
#define WORK_UNIT_SIZE (1ULL << 23ULL)
#endif

#ifndef WORK_RANDOM_SIZE
#define WORK_RANDOM_SIZE (WORK_UNIT_SIZE + 1000)
#endif

#ifndef BLOCK_SIZE
#define BLOCK_SIZE 1024ULL
#endif

#ifndef GPU_COUNT
#define GPU_COUNT 1ULL
#endif

#ifndef OFFSET
#define OFFSET 0
#endif

#ifndef END
#define END (1ULL << 48ULL)
#endif


__device__ inline int8_t extract(const int8_t heightMap[], uint32_t i) {
    return (int8_t) (heightMap[i >> 1ULL] >> ((i & 1ULL) << 2ULL)) & 0xF;
}

__device__ inline void increase(int8_t heightMap[], uint32_t i) {
    heightMap[i >> 1ULL] += 1ULL << ((i & 1ULL) << 2ULL);
}

namespace java_random {

    // Random::next(bits)
    __device__ inline uint32_t next(uint64_t *random, size_t *i, int32_t bits) {
        return (uint32_t) (random[++*i] >> (48ULL - bits));
    }

    __device__ inline int32_t next_int_unknown(uint64_t *random, size_t *i, int16_t bound) {
        if ((bound & -bound) == bound) {
            return (int32_t) ((bound * (random[++*i] >> 17ULL)) >> 31ULL);
        }

        int32_t bits, value;
        do {
            bits = random[++*i] >> 17ULL;
            value = bits % bound;
        } while (bits - value + (bound - 1) < 0);
        return value;
    }

    // Random::nextInt(bound)
    __device__ inline uint32_t next_int(uint64_t *random, size_t *i) {
        return java_random::next(random, i, 31) % 3;
    }

}

struct rand_params {
	uint64_t multiplier;
	uint64_t addend;
};

__host__ uint64_t get_start_seed(uint64_t offset) {
	uint64_t seed = 0;
	uint64_t i = 0;
	for (; i + 1024 * 1024 <= offset; i += 1024 * 1024) {
		seed = (seed * 280364631195649ULL + 215216710221824ULL) & RANDOM_MASK;
	}
	for (; i < offset; i++) {
		seed = (seed * RANDOM_MULTIPLIER + RANDOM_ADDEND) & RANDOM_MASK;
	}
	return seed;
}

__host__ void gen_rand_params(rand_params *rp, size_t n) {
	rp[0].multiplier = 1;
	rp[0].addend = 0;
	for (size_t i = 1; i < n; i++) {
		rp[i].multiplier = (rp[i - 1].multiplier * RANDOM_MULTIPLIER) & RANDOM_MASK;
		rp[i].addend = (rp[i - 1].addend * RANDOM_MULTIPLIER + RANDOM_ADDEND) & RANDOM_MASK;
	}
}

// TODO: generate low bits seperately so crack() only has to index 32 bit values, ~3% speed improvement
__global__ __launch_bounds__(BLOCK_SIZE, 2) void init(rand_params *rp, uint64_t *random, uint64_t seed, uint64_t m, uint64_t a) {
	size_t index = blockIdx.x * blockDim.x + threadIdx.x;
	size_t stride = blockDim.x * gridDim.x;
	seed = (seed * rp[index].multiplier + rp[index].addend) & RANDOM_MASK;
	random[index] = seed;
	for (size_t i = index + stride; i < WORK_RANDOM_SIZE; i += stride) {
		seed = (seed * m + a) & RANDOM_MASK;
		random[i] = seed;
	}
}

__global__ __launch_bounds__(BLOCK_SIZE, 2) void crack(uint64_t *random, int32_t *num_seeds, uint64_t *seeds) {
    size_t seedIndex = blockIdx.x * blockDim.x + threadIdx.x;
	uint64_t originalSeed = random[seedIndex];

    int8_t heightMap[512];

#pragma unroll
    for (int i = 0; i < 512; i++) {
        heightMap[i] = 0;
    }

    int16_t currentHighestPos = 0;
    int16_t terrainHeight;
    int16_t initialPosX, initialPosY, initialPosZ;
    int16_t posX, posY, posZ;
    int16_t offset, posMap;

    int16_t i, a, j;

    for (i = 0; i < 10; i++) {
        // Keep, most threads finish early this way
        if (WANTED_CACTUS_HEIGHT - extract(heightMap, currentHighestPos) > 9 * (10 - i))
            return;

        initialPosX = java_random::next(random, &seedIndex, 4) + 8;
        initialPosZ = java_random::next(random, &seedIndex, 4) + 8;
        terrainHeight = (extract(heightMap, initialPosX + initialPosZ * 32) + FLOOR_LEVEL + 1) * 2;

        initialPosY = java_random::next_int_unknown(random, &seedIndex, terrainHeight);

        for (a = 0; a < 10; a++) {
            posX = initialPosX + java_random::next(random, &seedIndex, 3) - java_random::next(random, &seedIndex, 3);
            posY = initialPosY + java_random::next(random, &seedIndex, 2) - java_random::next(random, &seedIndex, 2);
            posZ = initialPosZ + java_random::next(random, &seedIndex, 3) - java_random::next(random, &seedIndex, 3);

            posMap = posX + posZ * 32;
            // Keep
            if (posY <= extract(heightMap, posMap) + FLOOR_LEVEL && posY >= 0)
                continue;

            offset = 1 + java_random::next_int_unknown(random, &seedIndex, java_random::next_int(random, &seedIndex) + 1);

            for (j = 0; j < offset; j++) {
                if ((posY + j - 1) > extract(heightMap, posMap) + FLOOR_LEVEL || posY < 0) continue;
                if ((posY + j) <= extract(heightMap, (posX + 1) + posZ * 32) + FLOOR_LEVEL && posY >= 0) continue;
                if ((posY + j) <= extract(heightMap, posX + (posZ - 1) * 32) + FLOOR_LEVEL && posY >= 0) continue;
                if ((posY + j) <= extract(heightMap, (posX - 1) + posZ * 32) + FLOOR_LEVEL && posY >= 0) continue;
                if ((posY + j) <= extract(heightMap, posX + (posZ + 1) * 32) + FLOOR_LEVEL && posY >= 0) continue;

                increase(heightMap, posMap);

                if (extract(heightMap, currentHighestPos) < extract(heightMap, posMap)) {
                    currentHighestPos = posMap;
                }
            }
        }

        if (extract(heightMap, currentHighestPos) >= WANTED_CACTUS_HEIGHT) {
            seeds[atomicAdd(num_seeds, 1)] = originalSeed;
            return;
        }
    }
}


struct GPU_Node {
    int *num_seeds;
    uint64_t *seeds;
	uint64_t *random;
	rand_params *rp;
};

void setup_gpu_node(GPU_Node *node, int32_t gpu) {
    hipSetDevice(gpu);
    hipMallocManaged(&node->num_seeds, sizeof(*node->num_seeds));
    hipMallocManaged(&node->seeds, 1ULL << 10ULL);
    hipMallocManaged(&node->random, WORK_RANDOM_SIZE * sizeof(*node->random));
    hipMallocManaged(&node->rp, WORK_RANDOM_SIZE * sizeof(*node->rp));
}

GPU_Node nodes[GPU_COUNT];
uint64_t offset = OFFSET;
uint64_t seed = get_start_seed(OFFSET);
uint64_t count = 0;
std::mutex info_lock;
std::vector<uint64_t> seeds;
rand_params *rp;

void gpu_manager(int32_t gpu_index) {
    std::string fileName = "kaktoos_seeds" + std::to_string(gpu_index) + ".txt";
    FILE *out_file = fopen(fileName.c_str(), "w");
	hipSetDevice(gpu_index);
	for (size_t i = 0; i < WORK_RANDOM_SIZE; i++)
		nodes[gpu_index].rp[i] = rp[i];
    while (offset < END) {
        *nodes[gpu_index].num_seeds = 0;
		uint64_t m = rp[WORK_UNIT_SIZE].multiplier, a = rp[WORK_UNIT_SIZE].addend;
        init<<<WORK_UNIT_SIZE / BLOCK_SIZE, BLOCK_SIZE, 0>>>(nodes[gpu_index].rp, nodes[gpu_index].random, seed, m, a);
        info_lock.lock();
		seed = (seed * m + a) & RANDOM_MASK;
		offset += WORK_UNIT_SIZE;
        info_lock.unlock();
        hipDeviceSynchronize();
        crack<<<WORK_UNIT_SIZE / BLOCK_SIZE, BLOCK_SIZE, 0>>>(nodes[gpu_index].random, nodes[gpu_index].num_seeds, nodes[gpu_index].seeds);
        hipDeviceSynchronize();
        for (int32_t i = 0, e = *nodes[gpu_index].num_seeds; i < e; i++) {
            fprintf(out_file, "%lld\n", (long long int) nodes[gpu_index].seeds[i]);
            seeds.push_back(nodes[gpu_index].seeds[i]);
        }
        fflush(out_file);
        info_lock.lock();
        count += *nodes[gpu_index].num_seeds;
        info_lock.unlock();
    }
    fclose(out_file);
}


int main() {
	printf("Searching %ld total seeds...\n", (long int) (END - OFFSET));
	
	rp = new rand_params[WORK_RANDOM_SIZE];
	gen_rand_params(rp, WORK_RANDOM_SIZE);
    std::thread threads[GPU_COUNT];

    time_t startTime = time(nullptr), currentTime;
    for (int32_t i = 0; i < GPU_COUNT; i++) {
        setup_gpu_node(&nodes[i], i);
        threads[i] = std::thread(gpu_manager, i);
    }

    using namespace std::chrono_literals;

    while (offset < END) {
        time(&currentTime);
        int timeElapsed = (int) (currentTime - startTime);
        double speed = (double) (offset - OFFSET) / (double) timeElapsed / 1000000.0;
        printf("Searched %lld seeds, offset: %lld found %lld matches. Time elapsed: %ds. Speed: %.2fm seeds/s. %f%%\n",
               (long long int) (offset - OFFSET),
               (long long int) offset,
               (long long int) count,
               timeElapsed,
               speed,
               (double) (offset - OFFSET) / (END - OFFSET) * 100);

        if (timeElapsed % 2000 == 0) {
            printf("Backup seed list:\n");
            for (auto &seed : seeds) {
                printf("%llu\n", (unsigned long long) seed);
            }
        }

        std::this_thread::sleep_for(1s);
    }

    for (auto &thread : threads) {
        thread.join();
    }

    printf("Done!\n");
    printf("But, verily, it be the nature of dreams to end.\n");
}